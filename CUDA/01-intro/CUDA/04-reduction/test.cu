
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel() {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  printf("Hello from thread.\n");
}



int main(){
  int host_a, host_b, host_c;
  int *dev_a, *dev_b, *dev_c;

  int size = sizeof (int);

  hipMalloc((void**) &dev_a, size);
  hipMalloc((void**) &dev_b, size);
  hipMalloc((void**) &dev_c, size);

  host_a = 2;
  host_b = 7;

  hipMemcpy(dev_a, &host_a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, &host_b, size, hipMemcpyHostToDevice);

  kernel <<< 1, 1 >>> ();

  hipDeviceSynchronize();

  hipMemcpy(&host_c, dev_c, size, hipMemcpyDeviceToHost);

  printf("C = %d \n", host_c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  printf("Hello, CUDA! \n");
}
